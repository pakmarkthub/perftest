
#include <hip/hip_runtime.h>
#include <stdint.h>


template <typename T>
__global__
void bufcmp(size_t buf_size, int *result, T *buf, T *cmp, size_t cmp_len, int is_lat)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < buf_size / sizeof(*buf)) {
		if (buf[i] != cmp[i % (cmp_len / sizeof(*buf))]) {
			*result = 1;
		}
		/* poison 8 bytes of the buffers every 64K. (GPU BAR1 mapping granularity) */
		if (i % (65536 / sizeof(*buf)) < 8 / sizeof(*buf)) {
			if (is_lat == 1) {
				/* In the latency case, clear the buffer that was written into */
				buf[i] = 0;
			} else {
				/* In the BW case, clear the verification buffer
				 * and rotate the bytes in the write buffer to
				 * avoid masking failures between iterations.
				 */
				cmp[i] = 0;
				if (i < (buf_size / sizeof(*buf)) - 1) {
					buf[i] = buf[(i + 1) % buf_size];
				}
			}
		}
	}
}

/*
 * Launch the kernel for comparing a GPU buffer vs. another expected/derived value.
 *
 * args:
 * buf - In the BW case, where we do client-side validation, this is the client-side buffer we write from. In the LAT case
 * this is the part of the buffer which gets written into. IN both cases, this value corresponds to part, or all of the buf
 * value in the workload's pingpong_context struct.
 * buf_size - the length of buf to compare.
 * cmp - In the BW case, where we do client-side validation, this is the client side buffer we read into from the remote
 * server buffer we just wrote from. It is the same size as buf. In the LAT case, it is a repeating byte pattern defined
 * at startup and of a much shorter length than the buffer.
 * cmp_len - The length of cmp to compare. This is assumed to be <= buf_size.
 * is_lat - flag to indicate if we are in a LAT or BW workload. Affects the way we poison the buffers.
 *
 */
extern "C" void start_bufcmp_gpu_async(char *buf, size_t buf_size, char *cmp, size_t cmp_len, int *result, int is_lat)
{
	size_t num_iterations;

	/* If both buffers don't have a length that is an even multiple of 8 bytes, fall back to a bytewise comparison. */
	if (buf_size < 8 || buf_size % sizeof(uint64_t) != 0 || cmp_len % sizeof(uint64_t) != 0) {
			bufcmp<<<(buf_size + 1023) / 1024, 1024>>>(buf_size, result, buf, cmp, cmp_len, is_lat);
	} else {
		num_iterations = buf_size / sizeof(uint64_t);
		bufcmp<<<(num_iterations + 1023) / 1024, 1024>>>(buf_size, result, (uint64_t *)buf, (uint64_t *)cmp, cmp_len, is_lat);
	}

}
